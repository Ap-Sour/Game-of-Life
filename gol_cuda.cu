#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>

#define threads_per_block 256

__global__ void run(int rows, int cols, int* board, int* next)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int neighbors = 0;
	if(i < rows*cols)
	{
		if((i == 0) || (i%cols == 0))
		{
			if(i == 0)
			{
				neighbors = board[1] + board[cols-1] + board[cols] + board[(rows-1)*cols] + board[cols+1] + board[(rows*cols)-1] + board[(2*cols)-1] + board[(rows-1)*cols +1];
			}
			else if(i == ((rows-1)*cols))
			{
				neighbors = board[i+1] + board[rows*cols -1] + [(rows-2)*cols] + board[0] + board[(rows-2)*cols +1] + board[cols-1] + board[1] + board[i-1];
			}
			else
			{
				neighbors = board[i+1] + board[i+cols-1] + board[i-cols] + board[i+cols] + board[i+cols+1] + board[i-1] + board[i-1+2*cols] + board[i-cols+1];
			}
		}
		else if((i+1)%cols == 0)
		{
			if(i == (cols-1))
			{
				neighbors = board[i-1] + board[i-cols+1] + board[rows*cols -1] + board[i+cols] + board[i+cols-1] + board[(rows-1)*cols] + board[rows*cols -2] + board[i+1];
			}
			else if(i == (rows*cols -1))
			{
				neighbors = board[i-1] + board[i-cols+1] + board[i-cols] + board[cols-1] + board[cols-2] + board[i+1-2*cols] + board[i-cols-1] + board[0];
			}
			else
			{
				neighbors = board[i-1] + board[i-cols+1] + board[i-cols] + board[i+cols] + board[i+cols-1] + board[i+1-2*cols] + board[i-cols-1] + board[i+1];
			}
		}
		else
		{
			neighbors = board[i+1] + board[i-1] + board[i-cols] + board[i+cols] + board[i+cols-1] + board[i+cols+1] + board[i-cols-1] + board[i-cols+1];
		}
		if((board[i] == 1) && (neighbors<2 || neighbors>3)) next[i] = 0;
		else if((board[i] == 0) && (neighbors == 3)) next[i] = 1;
		else next[i] = board[i];
	}
}

int continue_running(int *board, int *next, int* same, int* empty, int rows, int cols)
{
	int i;
	int same = 1;
	int sum = 0;
	for(i = 0; i < rows*cols; i++)
	{
		sum = sum + next[i];
		if(board[i] != next[i])
			same = 0;
	}
	if((sum > 0) && (!same)) return 1;
	else return 0;
}

int main(int argc, char* argv[])
{
	int *board, *next;
	int *d_board, *d_next;
	int M, N, gens, i;
	N = strtol(argv[1], NULL, 10);
	gens = strtol(argv[2], NULL, 10);
	
	M = N*N;
	board = (int *)malloc(M*sizeof(int));
	next = (int *)malloc(M*sizeof(int));

	hipMalloc(&d_board, M*sizeof(int));
	hipMalloc(&d_next, M*sizeof(int));

	for(i = 0; i < M; ++i )
	{
		srand(time(NULL)); 
		board[i] = rand()%2;
		next[i] = 0;
	}
	int *temp;
	for(i = 0; i < gens; i++)
	{
		hipMemcpy(d_board, board, M*sizeof(int), hipMemcpyHostToDevice );
		hipMemcpy(d_next, next, M*sizeof(int), hipMemcpyHostToDevice );
		
		run<<(N+(threads_per_block-1))/threads_per_block, threads_per_block>>(N, N, d_board, d_next);
		
		hipMemcpy(board, d_board, M*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(next, d_next, M*sizeof(int), hipMemcpyDeviceToHost);
		if(continue_running(board, next, N, N))
		{
			temp = board;
			board = next;
			next = temp;
		}
		else break;
	}
	
	hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

	free(board);
	free(next);

	hipFree(d_board);
	hipFree(d_next);

	return 0;
}
